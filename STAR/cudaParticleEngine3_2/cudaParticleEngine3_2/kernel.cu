#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
//#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
//#include "xnaTypes.h"
#include "particleSystemTypes.h"
void __syncthreads();
void checkCUDAError(const char *msg);

__global__ void doParticleCalculations(Vector2 *positions,Vector2 *velocities,float *lifetimes,struct Rectangle* rects,unsigned char *alphas,float *Rotations,int N,struct ParticleOptions options,struct Rectangle* collisionRects,int nColRects)
{

    Vector2 grav;
    Vector2 newpos;
    //(float)gametime.ElapsedGameTime.TotalSeconds;
	float elapsed = options.elapsedGameTime;
    //float elapsed = 0.003f;
    //int length = particles.GetLength(0);
	//particles[0].lifetime = thread_start_idx;
	int inOffset  = blockDim.x * blockIdx.x;
	int i = inOffset + threadIdx.x;
	//for (int i = blockDim.x * blockIdx.x; i < thread_end_idx; i+=STRIDE)
	if(i<N)
    {
		__syncthreads();
		//device_particles[i].lifetime = elapsed;
		//particles[i].lifetime += 10;
		//if (lifetimes[i] < options.totalLifeTime)
        {
			
            if (options.gravityType == OverallForce)
            {
                grav = options.gravity;
            }
			else if (options.gravityType == Newton)
            {
				grav= Vector2();
				for(int massIndex = 0; massIndex < options.newtonMassSize;massIndex++)
                {
					grav += ((options.mass[massIndex].center - positions[i]) * (options.mass[massIndex].weight / ((options.mass[massIndex].center - positions[i]).LengthSquared())));
                }
                //grav /= 2;
            }
            else
            {
                grav = (options.gravity - positions[i]) * (1000 / (options.gravity - positions[i]).Length());
            }
            velocities[i] += grav * elapsed;
            velocities[i] *= options.AirFriction;
            newpos = positions[i] + velocities[i] * elapsed;
            if (options.collisionType == Collision)
            {
				for (int collisionIndex = 0; collisionIndex< nColRects;collisionIndex++)
                {
                    //RightCollision
                    if (positions[i].X + options.size <= collisionRects[collisionIndex].X && newpos.X + options.size >= collisionRects[collisionIndex].X)
                    {
                        if (newpos.Y + options.size >= collisionRects[collisionIndex].Y && newpos.Y <= collisionRects[collisionIndex].Bottom())
                        {
                            velocities[i].X /= -1;//+ (float)(rand.NextDouble() / 10);
                            velocities[i] /= options.Friction;
                            newpos = positions[i] + velocities[i] * elapsed;
                            continue;
                        }
                    }

                    //LeftCollision
                    else if (positions[i].X > collisionRects[collisionIndex].Right() && newpos.X <= collisionRects[collisionIndex].Right())
                    {
                        if (newpos.Y + options.size >= collisionRects[collisionIndex].Y && newpos.Y <= collisionRects[collisionIndex].Bottom())
                        {
                            velocities[i].X /= -1;//+ (float)(rand.NextDouble() / 10);
                            velocities[i] /= options.Friction;
                            newpos = positions[i] + velocities[i] * elapsed;
                            continue;

                        }
                    }

                    //BottomCollision
                    else if (positions[i].Y + options.size <= collisionRects[collisionIndex].Y && newpos.Y + options.size >= collisionRects[collisionIndex].Y)
                    {
                        if (positions[i].X + options.size >= collisionRects[collisionIndex].X && positions[i].X <= collisionRects[collisionIndex].Right())
                        {
                            velocities[i].Y /= -1;//+ (float)(rand.NextDouble() / 10);
                            velocities[i] /= options.Friction;
                            newpos = positions[i] + velocities[i] * elapsed;
                            continue;
                        }
                    }
                    //UpCollision
                    else if (positions[i].Y >= collisionRects[collisionIndex].Bottom() && newpos.Y <= collisionRects[collisionIndex].Bottom())
                    {
                        if (newpos.X + options.size >= collisionRects[collisionIndex].X && newpos.X <= collisionRects[collisionIndex].Right())
                        {
                            velocities[i].Y /= -1;//+ (float)(rand.NextDouble() / 10);
                            velocities[i] /= options.Friction;
                            newpos = positions[i] + velocities[i] * elapsed;
                            continue;
                        }
                    }


                }
            }

			lifetimes[i] = lifetimes[i] + options.elapsedGameTime;
            Vector2 newVelocity =velocities[i] * elapsed;
			positions[i] += newVelocity;
			rects[i].X = (int)positions[i].X + options.size/2;
			rects[i].Y = (int)positions[i].Y + options.size / 2;
            //if ((newVelocity).LengthSquared() > 1)
            
			Rotations[i] = (float)acos(velocities[i].X / (velocities[i].Length()));
                if (velocities[i].Y < 0)
                    Rotations[i] *= -1;
            
			alphas[i] = (char)(255 - (255 * (lifetimes[i] / options.totalLifeTime)));

        }
		
    }
            
            //updatethread.Abort();
        
        //Thread.Sleep(0);
}


/*Particle *device_particles;
Vector2 *device_positions,*device_velocities;
struct Rectangle *device_rects,*device_collisionrects;
float *device_lifetimes,*device_Rotations;
unsigned char* device_alphas;
int numberColRects;*/
int main()
{

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
	//if(CudaAvailable())
	{
		printf("success");
	}
	return 0;
}

extern "C"

{

	//from deviceQuery Example in CUDA Toolkit 3.2
	__declspec (dllexport) bool CudaAvailable()
	{
        
    int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		return false;
	}

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
        return false;

    int dev;   
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
			// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                return false;
			else
                return true;
                
        }
	}
	return true;
	}

	__declspec(dllexport) struct ArrayHolder* CudaInitialize(Vector2 *positions,Vector2 *velocities,float *Rotations,float *lifetimes,struct Rectangle * rects,unsigned char *alphas, int N)
	{
		struct ArrayHolder *holder;
		holder = (struct ArrayHolder *) malloc(sizeof(ArrayHolder));
		size_t sizeVector = N*8;
		size_t sizeFloat = N * sizeof(float);
		size_t sizeRect = N * sizeof(struct Rectangle);
		size_t sizeChar = N* sizeof(unsigned char);


		if(hipMalloc((void**) &(holder->device_Rotations), sizeFloat) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMalloc((void**) &(holder->device_lifetimes), sizeFloat) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMalloc((void**) &(holder->device_rects), sizeRect) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMalloc((void**) &holder->device_alphas, sizeChar) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMalloc((void**) &holder->device_positions, sizeVector) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMalloc((void**) &holder->device_velocities, sizeVector) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}

		if(hipMemcpy(holder->device_lifetimes, lifetimes, sizeFloat, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 11\n");
		}

		if(hipMemcpy(holder->device_rects, rects, sizeRect, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_Rotations, Rotations, sizeFloat, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_alphas, alphas, sizeChar, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_positions, positions, sizeVector, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}


		if(hipMemcpy(holder->device_velocities, velocities, sizeVector, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		return holder;

	}


	
	__declspec(dllexport) void CudaReset(struct ArrayHolder* holder,Vector2 *positions,Vector2 *velocities,float *Rotations,float *lifetimes,struct Rectangle * rects,unsigned char *alphas, int N)
	{
		size_t sizeVector = N*8;
		size_t sizeFloat = N * sizeof(float);
		size_t sizeRect = N * sizeof(struct Rectangle);
		size_t sizeChar = N* sizeof(unsigned char);

		if(hipMemcpy(holder->device_lifetimes, lifetimes, sizeFloat, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 2\n");
		}

		if(hipMemcpy(holder->device_rects, rects, sizeRect, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_Rotations, Rotations, sizeFloat, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_alphas, alphas, sizeChar, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

		if(hipMemcpy(holder->device_positions, positions, sizeVector, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}


		if(hipMemcpy(holder->device_velocities, velocities, sizeVector, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}

	}

	__declspec(dllexport) void CudaCollisionRectanglesChanged(struct ArrayHolder *holder,struct Rectangle* collisionRectangles,int Size)
	{
		size_t collisionRectsSize = Size * sizeof(struct Rectangle);
		//if(holder->device_collisionrects != NULL)
		//	hipFree(holder->device_collisionrects);
		if(hipMalloc((void**) &holder->device_collisionrects, collisionRectsSize) != hipSuccess)
		{
			printf("hipMalloc Error!\n");
		}
		if(hipMemcpy(holder->device_collisionrects, collisionRectangles, collisionRectsSize, hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error 1\n");
		}
		holder->numberColRects = Size;
	}

	__declspec(dllexport) void SinglePArticleChanged(struct ArrayHolder *holder,Vector2 positions,Vector2 velocities,float Rotations,float lifetimes,struct Rectangle  rects,unsigned char alphas,int number)
	{
		if(hipMemcpy(&(holder->device_velocities[number]),&velocities,sizeof(Vector2),hipMemcpyHostToDevice) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error Resetting Particle");
		}
		hipMemcpy(&(holder->device_positions[number]),&positions,sizeof(Vector2),hipMemcpyHostToDevice);
		hipMemcpy(&(holder->device_Rotations[number]),&Rotations,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(&(holder->device_lifetimes[number]),&lifetimes,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(&(holder->device_alphas[number]),&alphas,sizeof(char),hipMemcpyHostToDevice);
		hipMemcpy(&(holder->device_rects[number]),&rects,sizeof(struct Rectangle),hipMemcpyHostToDevice);
		/*[number] = positions;
		device_velocities[number] = velocities;
		device_Rotations[number] = Rotations;
		device_lifetimes[number] = lifetimes;
		device_alphas[number] = alphas;
		device_rects[number]= rects;*/

	}

	__declspec(dllexport) void CudaMain(struct ArrayHolder*holder,Vector2 *positions,Vector2 *velocities,float *Rotations,float *lifetimes,struct Rectangle * rects,unsigned char *alphas, int N,ParticleOptions options)

	{
		size_t sizeVector = N*8;
		size_t sizeFloat = N * sizeof(float);
		size_t sizeRect = N * sizeof(struct Rectangle);
		size_t sizeChar = N* sizeof(unsigned char);
		//if(hipMemcpy(device_velocities, velocities, sizeVector, hipMemcpyHostToDevice) != hipSuccess)

		//{

		//	printf("Error 1\n");

		//}


		//if(hipMemcpy(device_positions, positions, sizeVector, hipMemcpyHostToDevice) != hipSuccess)

		//{

		//	printf("Error 1\n");

		//}

		//particles[0].lifetime = 1337;
		//if(hipMemcpy(device_particles, particles, size, hipMemcpyHostToDevice) != hipSuccess)

		//{

		//	printf("Error 2\n");

		//}
		//particles[0].lifetime = 1338;
		int numThreadsPerBlock = 256;
		int numBlocks = N/numThreadsPerBlock+ (N%numThreadsPerBlock == 0?0:1);
		dim3 dimGrid(numBlocks);
		dim3 dimBlock(numThreadsPerBlock);
		doParticleCalculations <<< dimGrid, dimBlock >>> (holder->device_positions,holder->device_velocities,holder->device_lifetimes,holder->device_rects,holder->device_alphas,holder->device_Rotations, N,options,holder->device_collisionrects,holder->numberColRects);
		hipDeviceSynchronize();
		//checkCUDAError("Error 2222222");
		if((hipMemcpy(positions, holder->device_positions, sizeVector, hipMemcpyDeviceToHost)) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333331");
		}

		if((hipMemcpy(rects, holder->device_rects, sizeRect, hipMemcpyDeviceToHost)) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333332");
		}

		if((hipMemcpy(lifetimes, holder->device_lifetimes, sizeFloat, hipMemcpyDeviceToHost)) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333333");
		}

		if((hipMemcpy(Rotations, holder->device_Rotations, sizeFloat, hipMemcpyDeviceToHost)) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333334");
		}

		if((hipMemcpy(alphas, holder->device_alphas, sizeChar, hipMemcpyDeviceToHost)) != hipSuccess)
		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333335");
		}

		if((hipMemcpy(velocities, holder->device_velocities, sizeVector, hipMemcpyDeviceToHost)) != hipSuccess)

		{
			hipDeviceSynchronize();
			checkCUDAError("Error 333336");
		}

	}


	
	__declspec(dllexport) bool Clear(struct ArrayHolder* holder)
	{
		bool success = true;
		if(hipFree(holder->device_positions)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_velocities)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_rects)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_lifetimes)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_Rotations)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_collisionrects)!=hipSuccess)
			success=false;
		if(hipFree(holder->device_alphas)!=hipSuccess)
			success=false;

		return success;
	}
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        //exit(EXIT_FAILURE);
    }                         
}